#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include "tanhshrinkPlugin.h"
#include <hip/hip_fp16.h>
#include <cmath>

template <typename T_DATA>
     __global__ void kernelTanhshrink(
         T_DATA* inputs,
         T_DATA* outputs,
         int N
         ){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N){
         outputs[index] = inputs[index] - tanh(inputs[index]);
    }
}

 template <typename T>
 int inferenceTanhshrink(
     int batchSize,
     int iC,
     int iH,
     int iW,
     int oC,
     int oH,
     int oW,
     T* inputs,
     T* outputs,
     hipStream_t stream){
     // NCHW
     const int nThreads = 512;
     int size = batchSize * iC * iH * iW;

     int nBlocks = (int)((float) size / nThreads) + 1;

     kernelTanhshrink <<< nBlocks, nThreads, 0, stream >>> (inputs, outputs, size);
     hipDeviceSynchronize();

     hipError_t err = hipGetLastError();
     if ( hipSuccess != err )
     {
         fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 __FILE__, __LINE__, hipGetErrorString( err ) );
         return 1;
     }
     return 0;
 }

 int TanhshrinkPlugin::enqueue(
     int batchSize, const void* const* inputs, void** outputs, void* workspace, hipStream_t stream)
 {
     switch(iType){
         case DataType::kFLOAT:
             return inferenceTanhshrink(batchSize, iC, iH, iW, oC, oH, oW, (float*)inputs[0], (float*)outputs[0], stream);
     }
     return 1;
 }
